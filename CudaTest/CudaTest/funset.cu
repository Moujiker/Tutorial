#include "hip/hip_runtime.h"
#include "funset.cuh"
#include <stdio.h>

__global__ void add(int a, int b, int* c)
{
	*c = a + b;
}

//__global__���������ϵ��ò����豸������
__global__ void add_blockIdx(int* a, int* b, int* c)
{
	//�����������������
	//����blockIdx����һ�����ñ�������CUDA����ʱ���Ѿ�Ԥ�ȶ������������
	//�˱����а�����ֵ���ǵ�ǰִ���豸������߳̿������
	int tid = blockIdx.x;//this thread handles the data at its thread id
	if (tid < NUM)
		c[tid] = a[tid] + b[tid];
}

//__device__����ʾ���뽫��GPU���������������У�
//���ڴ˺���������Ϊ__device__���������ֻ�ܴ�����__device__��������
//��__global__�����е�������
__device__ int julia(int x, int y) 
{
	const float scale = 1.5;
	float jx = scale * (float)(DIM/2 - x)/(DIM/2);
	float jy = scale * (float)(DIM/2 - y)/(DIM/2);

	hipComplex c(-0.8, 0.156);
	hipComplex a(jx, jy);

	int i = 0;
	for (i=0; i<200; i++) {
		a = a * a + c;

		if (a.magnitude2() > 1000)
			return 0;
	}

	return 1;
}

__global__ void kernel_julia(unsigned char *ptr)
{
	//map from blockIdx to pixel position
	int x = blockIdx.x;
	int y = blockIdx.y;
	//gridDimΪ���ñ����������е��߳̿���˵��gridDim��һ�����������������̸߳�ÿһά�Ĵ�С
	//�˴�gridDim��ֵ��(DIM, DIM)
	int offset = x + y * gridDim.x;

	//now calculate the value at that position
	int juliaValue = julia(x, y);

	ptr[offset*4 + 0] = 255 * juliaValue;
	ptr[offset*4 + 1] = 0;
	ptr[offset*4 + 2] = 0;
	ptr[offset*4 + 3] = 255;
}

__global__ void add_threadIdx(int* a, int* b, int* c)
{
	//ʹ���߳������������ݽ�����������ͨ���߳̿�����(blockIdx.x)
	int tid = threadIdx.x;

	if (tid < NUM)
		c[tid] = a[tid] + b[tid];
}

__global__ void add_blockIdx_threadIdx(int* a, int* b, int* c)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	if (tid == 0) {
		printf("blockDim.x = %d, gridDim.x = %d\n", blockDim.x, gridDim.x);
	}

	while (tid < NUM) {
		c[tid] = a[tid] + b[tid];

		tid += blockDim.x * gridDim.x;
	}
}

__global__ void ripple_kernel(unsigned char *ptr, int ticks)
{
	// map from threadIdx/BlockIdx to pixel position
	//���̺߳��߳̿������ӳ�䵽ͼ������
	//��x��y��ֵ�������Ի��Ӷ��õ�����������е�һ��ƫ��
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	// now calculate the value at that position
	//����һ����ʱ��仯����������"����"
	float fx = x - DIM/2;
	float fy = y - DIM/2;
	float d = sqrtf(fx * fx + fy * fy);
	unsigned char grey = (unsigned char)(128.0f + 127.0f * cos(d/10.0f - ticks/7.0f) / (d/10.0f + 1.0f)); 

	ptr[offset*4 + 0] = grey;
	ptr[offset*4 + 1] = grey;
	ptr[offset*4 + 2] = grey;
	ptr[offset*4 + 3] = 255;
}

__global__ void dot_kernel(float *a, float *b, float *c)
{
	//������һ�������ڴ滺��������������ÿ���̼߳���ļӺ�ֵ
	__shared__ float cache[threadsPerBlock];
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int cacheIndex = threadIdx.x;

	float temp = 0;
	while (tid < NUM) {
		temp += a[tid] * b[tid];
		tid += blockDim.x * gridDim.x;
	}

	//set the cache values
	cache[cacheIndex] = temp;

	//synchronize threads in this block
	//���߳̿��е��߳̽���ͬ��
	//���������ȷ���߳̿��е�ÿ���̶߳�ִ����__syncthreads()ǰ������󣬲Ż�ִ����һ�����
	__syncthreads();

	//for reductions(��Լ), threadsPerBlock must be a power of 2 because of the following code
	int i = blockDim.x/2;
	while (i != 0) {
		if (cacheIndex < i)
			cache[cacheIndex] += cache[cacheIndex + i];
		//��ѭ�������и����˹����ڴ����cache��������ѭ������һ�ε�����ʼ֮ǰ��
		//��Ҫȷ����ǰ�����������̵߳ĸ��²������Ѿ����
		__syncthreads();
		i /= 2;
	}

	if (cacheIndex == 0)
		c[blockIdx.x] = cache[0];
}

__global__ void julia_kernel(unsigned char *ptr)
{
	//map from threadIdx/BlockIdx to pixel position
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	__shared__ float shared[16][16];

	//now calculate the value at that position
	const float period = 128.0f;

	shared[threadIdx.x][threadIdx.y] = 255 * (sinf(x*2.0f*PI/ period) + 1.0f) *(sinf(y*2.0f*PI/ period) + 1.0f) / 4.0f;

	//removing this syncthreads shows graphically what happens
	//when it doesn't exist.this is an example of why we need it.
	__syncthreads();

	ptr[offset*4 + 0] = 0;
	ptr[offset*4 + 1] = shared[15 - threadIdx.x][15 - threadIdx.y];
	ptr[offset*4 + 2] = 0;
	ptr[offset*4 + 3] = 255;
}

__global__ void RayTracing_kernel(Sphere *s, unsigned char *ptr)
{
	//map from threadIdx/BlockIdx to pixel position
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;
	float ox = (x - DIM/2);
	float oy = (y - DIM/2);

	float r=0, g=0, b=0;
	float maxz = -INF;

	for (int i = 0; i < SPHERES; i++) {
		float n;
		float t = s[i].hit(ox, oy, &n);
		if (t > maxz) {
			float fscale = n;
			r = s[i].r * fscale;
			g = s[i].g * fscale;
			b = s[i].b * fscale;
			maxz = t;
		}
	} 

	ptr[offset*4 + 0] = (int)(r * 255);
	ptr[offset*4 + 1] = (int)(g * 255);
	ptr[offset*4 + 2] = (int)(b * 255);
	ptr[offset*4 + 3] = 255;
}

__global__ void RayTracing_kernel(unsigned char *ptr)
{
	//map from threadIdx/BlockIdx to pixel position
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;
	float ox = (x - DIM/2);
	float oy = (y - DIM/2);

	float r=0, g=0, b=0;
	float maxz = -INF;

	for(int i = 0; i < SPHERES; i++) {
		float n;
		float t = s[i].hit(ox, oy, &n);
		if (t > maxz) {
			float fscale = n;
			r = s[i].r * fscale;
			g = s[i].g * fscale;
			b = s[i].b * fscale;
			maxz = t;
		}
	} 

	ptr[offset*4 + 0] = (int)(r * 255);
	ptr[offset*4 + 1] = (int)(g * 255);
	ptr[offset*4 + 2] = (int)(b * 255);
	ptr[offset*4 + 3] = 255;
}

__global__ void Heat_blend_kernel(float *dst, bool dstOut)
{
	//map from threadIdx/BlockIdx to pixel position
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	int left = offset - 1;
	int right = offset + 1;
	if (x == 0) left++;
	if (x == DIM-1) right--; 

	int top = offset - DIM;
	int bottom = offset + DIM;
	if (y == 0) top += DIM;
	if (y == DIM-1) bottom -= DIM;

	float t, l, c, r, b;

	if (dstOut) {
		//tex1Dfetch�Ǳ��������ú��������豸�ڴ�ȡ����
		t = tex1Dfetch(texIn, top);
		l = tex1Dfetch(texIn, left);
		c = tex1Dfetch(texIn, offset);
		r = tex1Dfetch(texIn, right);
		b = tex1Dfetch(texIn, bottom);

	} else {
		t = tex1Dfetch(texOut, top);
		l = tex1Dfetch(texOut, left);
		c = tex1Dfetch(texOut, offset);
		r = tex1Dfetch(texOut, right);
		b = tex1Dfetch(texOut, bottom);
	}

	dst[offset] = c + SPEED * (t + b + r + l - 4 * c);
}

__global__ void blend_kernel(float *dst, bool dstOut)
{
	//map from threadIdx/BlockIdx to pixel position
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	float t, l, c, r, b;
	if (dstOut) {
		t = tex2D(texIn2, x, y-1);
		l = tex2D(texIn2, x-1, y);
		c = tex2D(texIn2, x, y);
		r = tex2D(texIn2, x+1, y);
		b = tex2D(texIn2, x, y+1);
	} else {
		t = tex2D(texOut2, x, y-1);
		l = tex2D(texOut2, x-1, y);
		c = tex2D(texOut2, x, y);
		r = tex2D(texOut2, x+1, y);
		b = tex2D(texOut2, x, y+1);
	}
	dst[offset] = c + SPEED * (t + b + r + l - 4 * c);
}

__global__ void Heat_copy_const_kernel(float *iptr)
{
	//map from threadIdx/BlockIdx to pixel position
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	float c = tex1Dfetch(texConstSrc, offset);
	if (c != 0)
		iptr[offset] = c;
}

__global__ void copy_const_kernel(float *iptr) 
{
	//map from threadIdx/BlockIdx to pixel position
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	float c = tex2D(texConstSrc2, x, y);
	if (c != 0)
		iptr[offset] = c;
}

void generate_frame_opengl(uchar4 *pixels, void*, int ticks)
{
	dim3 grids(DIM / 16, DIM / 16);
	dim3 threads(16, 16);
	ripple_kernel_opengl<<<grids, threads>>>(pixels, ticks);
}

__global__ void ripple_kernel_opengl(uchar4 *ptr, int ticks)
{
	//map from threadIdx/BlockIdx to pixel position
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	// now calculate the value at that position
	float fx = x - DIM / 2;
	float fy = y - DIM / 2;
	float d = sqrtf(fx * fx + fy * fy);
	unsigned char grey = (unsigned char)(128.0f + 127.0f * cos(d/10.0f - ticks/7.0f) / (d/10.0f + 1.0f));    
	ptr[offset].x = grey;
	ptr[offset].y = grey;
	ptr[offset].z = grey;
	ptr[offset].w = 255;
}

__global__ void Heat_blend_kernel_opengl(float *dst, bool dstOut)
{
	//map from threadIdx/BlockIdx to pixel position
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	int left = offset - 1;
	int right = offset + 1;
	if (x == 0) left++;
	if (x == DIM-1) right--; 

	int top = offset - DIM;
	int bottom = offset + DIM;
	if (y == 0) top += DIM;
	if (y == DIM-1) bottom -= DIM;

	float t, l, c, r, b;
	if (dstOut) {
		t = tex1Dfetch(texIn, top);
		l = tex1Dfetch(texIn, left);
		c = tex1Dfetch(texIn, offset);
		r = tex1Dfetch(texIn, right);
		b = tex1Dfetch(texIn, bottom);

	} else {
		t = tex1Dfetch(texOut, top);
		l = tex1Dfetch(texOut, left);
		c = tex1Dfetch(texOut, offset);
		r = tex1Dfetch(texOut, right);
		b = tex1Dfetch(texOut, bottom);
	}
	dst[offset] = c + SPEED * (t + b + r + l - 4 * c);
}

__global__ void Heat_copy_const_kernel_opengl(float *iptr)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	float c = tex1Dfetch(texConstSrc, offset);
	if (c != 0)
		iptr[offset] = c;
}

__global__ void histo_kernel(unsigned char *buffer, long size, unsigned int *histo)
{
	//clear out the accumulation buffer called temp since we are launched with 256 threads, 
	//it is easy to clear that memory with one write per thread
	__shared__  unsigned int temp[256]; //�����ڴ滺����
	temp[threadIdx.x] = 0;
	__syncthreads();

	//calculate the starting index and the offset to the next block that each thread will be processing
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	while (i < size) {
		atomicAdd(&temp[buffer[i]], 1);
		i += stride;
	}

	//sync the data from the above writes to shared memory then add the shared memory values to the values from
	//the other thread blocks using global memory atomic adds same as before, since we have 256 threads,
	//updating the global histogram is just one write per thread!
	__syncthreads();
	atomicAdd(&(histo[threadIdx.x]), temp[threadIdx.x]);
}

__global__ void singlestream_kernel(int *a, int *b, int *c)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < NUM) {
		int idx1 = (idx + 1) % 256;
		int idx2 = (idx + 2) % 256;
		float as = (a[idx] + a[idx1] + a[idx2]) / 3.0f;
		float bs = (b[idx] + b[idx1] + b[idx2]) / 3.0f;
		c[idx] = (as + bs) / 2;
	}
}

__global__ void dot_kernel(int size, float *a, float *b, float *c)
{
	__shared__ float cache[threadsPerBlock];
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int cacheIndex = threadIdx.x;

	float temp = 0;
	while (tid < size) {
		temp += a[tid] * b[tid];
		tid += blockDim.x * gridDim.x;
	}

	//set the cache values
	cache[cacheIndex] = temp;

	//synchronize threads in this block
	__syncthreads();

	//for reductions(��Լ), threadsPerBlock must be a power of 2 because of the following code
	int i = blockDim.x / 2;
	while (i != 0) {
		if (cacheIndex < i)
			cache[cacheIndex] += cache[cacheIndex + i];
		__syncthreads();
		i /= 2;
	}

	if (cacheIndex == 0)
		c[blockIdx.x] = cache[0];
}