#include "hip/hip_runtime.h"
#include <iostream>

#include "hip/hip_runtime.h"
#include ""

using namespace std;


__global__ void add(int a, int b, int *c)
{
	*c = a + b;
}



bool InitCUDA()
{
	int count;
	hipGetDeviceCount(&count);
	if (count == 0)
	{
		fprintf(stderr, "There is no device.\n");
		return false;
	}
	int i;
	for (i = 0; i < count; i++)
	{
		hipDeviceProp_t prop;
		if (hipGetDeviceProperties(&prop, i) == hipSuccess)
		{
			if (prop.major >= 1)
			{
				//ö����ϸ��Ϣ
				printf("Identify: %s\n", prop.name);
				printf("Host Memory: %d\n", prop.canMapHostMemory);
				printf("Clock Rate: %d khz\n", prop.clockRate);
				printf("Compute Mode: %d\n", prop.computeMode);
				printf("Device Overlap: %d\n", prop.deviceOverlap);
				printf("Integrated: %d\n", prop.integrated);
				printf("Kernel Exec Timeout Enabled: %d\n", prop.kernelExecTimeoutEnabled);
				printf("Max Grid Size: %d * %d * %d\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
				printf("Max Threads Dim: %d * %d * %d\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
				printf("Max Threads per Block: %d\n", prop.maxThreadsPerBlock);
				printf("Maximum Pitch: %d bytes\n", prop.memPitch);
				printf("Minor Compute Capability: %d\n", prop.minor);
				printf("Number of Multiprocessors: %d\n", prop.multiProcessorCount);
				printf("32bit Registers Availble per Block: %d\n", prop.regsPerBlock);
				printf("Shared Memory Available per Block: %d bytes\n", prop.sharedMemPerBlock);
				printf("Alignment Requirement for Textures: %d\n", prop.textureAlignment);
				printf("Constant Memory Available: %d bytes\n", prop.totalConstMem);
				printf("Global Memory Available: %d bytes\n", prop.totalGlobalMem);
				printf("Warp Size: %d threads\n", prop.warpSize);
				break;
			}
		}
	}
	if (i == count)
	{
		fprintf(stderr, "There is no device supporting CUDA.\n");
		return false;
	}
	hipSetDevice(i);

	printf("CUDA initialized.\n");
	getchar();

	return true;
}

int main(){

	
	system("pause");
	return 0;
}